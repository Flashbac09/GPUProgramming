#include <hip/hip_runtime.h>
#include <stdio.h>
//json defs
int main(int argc, char **argv) { 
    // define total data elements 
    int nElem = 1024;
    
    // define grid and block structure
    dim3 block (1024,2,5);
    dim3 grid ((nElem+block.x-1)/block.x); 
    printf("grid.x %d block.x %d \n",grid.x, block.x);

    // reset block
    block.x = 512;
    grid.x = (nElem+block.x-1)/block.x; 
    printf("grid.x %d block.x %d \n",grid.x, block.x);

    // reset block
    block.x = 256;
    grid.x = (nElem+block.x-1)/block.x; 
    printf("grid.x %d block.x %d \n",grid.x, block.x);

    // reset block
    block.x = 128;
    grid.x = (nElem+block.x-1)/block.x; 
    printf("grid.x %d block.x %d \n",grid.x, block.x);
    printf("%d %d %d %d",grid.y,grid.z,block.y,block.z);
    // reset device before you leave cudaDeviceReset();
    return(0);
}
