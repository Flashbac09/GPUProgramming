//std global index -done
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include<string.h>
#include<malloc.h>
#include<hip/hip_runtime.h>

#include<map>

using namespace std;

//1:
int n;
double *a,*b,*c;
double *dev_a,*dev_b,*dev_c;
dim3 Dg(3,2,2);
dim3 Db(4,5,6);
__global__ void kernel(int n,double* a,double* b,double* c)
{
    int tid=0,bid=0,bdm=0,idx=0;
    printf("%d %d %d %d\n",blockDim.x,blockDim.y,gridDim.x,gridDim.y);
    tid=threadIdx.x+threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
    bid=blockIdx.x+blockIdx.y*gridDim.x+blockIdx.z*gridDim.x*gridDim.y;
    bdm=blockDim.x*blockDim.y*blockDim.z;
    idx=tid+bid*bdm;
    if(idx<n)
    c[idx]=a[idx]+b[idx];
}

__host__ int main()
{
    scanf("%d",&n);

    //2:
    a=(double*)malloc(sizeof(double)*n);
    b=(double*)malloc(sizeof(double)*n);
    c=(double*)malloc(sizeof(double)*n);
    for(int i=0;i<n;i++)
    {
        a[i]=i+1;
        b[i]=(i*2.6)/(i+2);
    }
    hipMalloc((void**)&dev_a,sizeof(double)*n);
    hipMalloc((void**)&dev_b,sizeof(double)*n);
    hipMalloc((void**)&dev_c,sizeof(double)*n);

    //3:
    hipMemcpy(dev_a,a,sizeof(double)*n,hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,sizeof(double)*n,hipMemcpyHostToDevice);

    //4:
    //Dg.x=3;Dg.y=2;Dg.z=2;
    //Db.x=2;Db.y=1;Db.z=1;
    kernel<<<Dg,Db>>>(n,dev_a,dev_b,dev_c);

    //5:
    hipMemcpy(c,dev_c,sizeof(double)*n,hipMemcpyDeviceToHost);


    for(int i=0;i<n;i++)
    {
        printf("%lf ",c[i]);
    }

    //6:
    hipFree(dev_a);hipFree(dev_b);hipFree(dev_c);
    free(a);free(b);free(c);
    return 0;
}