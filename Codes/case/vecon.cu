#include "hip/hip_runtime.h"
//<<<cuda vecadd config>>>
//<<<thread:1D>>>

//Tips Updating:
//1:(void**)&dev for hipMalloc[no instance of overloaded function "hipMalloc" matches the argument list]
//2:don't define variables in main
//3:main a,b,c,without malloc,its length must be determined before using them(better define n)
#include<stdio.h>
#include<iostream>
#include<stdlib.h>
#include<malloc.h>
#include<math.h>
#include<hip/hip_runtime.h>
double *dev_a,*dev_b,*dev_c;
__global__ void vecadd(double* a,double* b,double* c)
{
    while(tid<n)
    {
        c[tid]=a[tid]+b[tid];
        tid+=blockDim.x*threadIdx.y+threadIdx.x;
    }
}

__host__ int main()
{
    int n;
    scanf("%d",&n);
    double a[n],b[n],c[n];
    memset(a,0,sizeof(double)*n);
    memset(b,0,sizeof(double)*n);
    memset(c,0,sizeof(double)*n);
    for(int i=0;i<n;i++)
    {
        scanf("%lf%lf",&a[i],&b[i]);
    }
    hipMalloc((void**)&dev_a,n*sizeof(double));
    hipMalloc((void**)&dev_b,n*sizeof(double));
    hipMalloc((void**)&dev_c,n*sizeof(double));
    hipMemcpy(dev_a,a,n*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,n*sizeof(double),hipMemcpyHostToDevice);
    vecadd<<<(n+127)/128,128>>>(dev_a,dev_b,dev_c);
    hipMemcpy(c,dev_c,n*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<n;i++)
    {
        printf("%lf\n",c[i]);
    }
    hipFree(dev_a);hipFree(dev_b);hipFree(dev_c);
    return 0;
}