#include<stdio.h>
#include<iostream>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
//vector add, given dimension n.
//#1dim block,1dim thread(int for <<<>>>)
//#set maxdimlimit
//#not once for all(means limitless vecdim)
//loop in threads
//SIZE:  2(1 for call once),2^31-1,512
#define n 2
double a[n],b[n],c[n];
//KEY:address for memcpy,set pointers.//
double *dev_a,*dev_b,*dev_c;


__global__ void vecadd(double *a,double *b,double *c)
{
    int index=blockDim.x*blockIdx.x+threadIdx.x;
    while(index<n)
    {
        c[index]=a[index]+b[index];
        index+=gridDim.x*blockDim.x;
    }
    
}


__host__ int main()
{
    //step 0 data
    for(int i=0;i<n;i++)
    scanf("%lf%lf",&a[i],&b[i]);
    //step 1 malloc for device
    hipMalloc((void**)&dev_a,n*sizeof(double));
    hipMalloc((void**)&dev_b,n*sizeof(double));
    hipMalloc((void**)&dev_c,n*sizeof(double));
    //step 2 data that device needs(H2D)
    hipMemcpy(dev_a,a,n*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,n*sizeof(double),hipMemcpyHostToDevice);
    //Which is Unnecessary:  cudaMemcpy(dev_c,c,n*sizeof(double),cudaMemcpyHostToDevice);
    //step 3 kernel function
    vecadd<<<128,128>>>(dev_a,dev_b,dev_c);
    //step 4 send result back to host(D2H)
    hipMemcpy(c,dev_c,n*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<n;i++)
    {
        printf("%lf + %lf = %lf\n",a[i],b[i],c[i]);
    }
    //step 5:CUDAFREE
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
