#include<stdio.h>
#include<iostream>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<malloc.h>

//doc prodcut,ultilizing shared memory
#define imin(a,b) (a<b?a:b)
const int N=3;
const int threadsPerBlock=256;
const int blocksPerGrid=imin(32,(N+threadsPerBlock-1)/threadsPerBlock);

__global__ void dot(double* a,double* b,double* c)
{
    __shared__ double cache[threadsPerBlock];
    int tindex=threadIdx.x+blockDim.x*blockIdx.x;
    int cacheindex=threadIdx.x;
    double temp=0;
    while(tindex<N)
    {
        temp+=a[tindex]+b[tindex];
        tindex+=blockDim.x*gridDim.x;
    }
    cache[cacheindex]=temp;//for sum
    __syncthreads();//threads sync
    //Reduction
    int i=blockDim.x/2;
    while(i!=0)
    {
        if(cacheindex<i)
        {
            cache[cacheindex]+=cache[cacheindex+i];
        }
        __syncthreads();
        i=i/2;
    }
    if(cacheindex==0)
        c[blockIdx.x]=cache[0];//to each block
}

__host__ int main()
{
    //1:list
    double *a,*b,c,*pc;
    double *dev_a,*dev_b,*dev_c,*dev_pc;
    //2.1:malloc CPU
    a=(double*)malloc(N*sizeof(double));
    b=(double*)malloc(N*sizeof(double));
    pc=(double*)malloc(blocksPerGrid*sizeof(double));
    //2.2:malloc GPU
    hipMalloc((void**)&dev_a,N*sizeof(double));
    hipMalloc((void**)&dev_b,N*sizeof(double));
    hipMalloc((void**)&dev_pc,blocksPerGrid*sizeof(double));
    for(int i=0;i<N;i++)
    {
        a[i]=i;b[i]=i*i;
    }
    //3.Memcpy forward
    hipMemcpy(dev_a,a,N*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,N*sizeof(double),hipMemcpyHostToDevice);
    //4.Kernel Function
    dot<<<blocksPerGrid,threadsPerBlock>>>(dev_a,dev_b,dev_pc);
    //5.Memcpy back
    hipMemcpy(pc,dev_pc,blocksPerGrid*sizeof(double),hipMemcpyDeviceToHost);
    c=0;
    for(int i=0;i<N;i++)
    {
        c+=pc[i];
    }
    printf("%lf",c);
    //6.1:cudafree
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_pc);
    //6.2:free
    free(a);free(b);free(pc);
    return 0;
}
