#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<string.h>
//device info code
/*
__global__ void Devcount()
{
    hipGetDeviceCount(&count);
}
*/
/*a numinfo for total available GPU
  use hipGetDeviceProperties to control props,which is a struct*/
__host__ int main()
{
    int count;
    hipDeviceProp_t p;
    hipGetDeviceCount(&count);
    printf("Devices Available:  %d\n",count);
    for(int i=0;i<count;i++)
    {
        hipGetDeviceProperties(&p,i);
    }
    //some examples,not full list of hipGetDeviceProperties struct.
    printf("DeviceName:  %s\n",p.name);
    printf("compute capability(major-minor): %d to %d\n",p.major,p.minor);
    printf("clockRate:  %d\n",p.clockRate);
    return 0;
}