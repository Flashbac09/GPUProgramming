#include <iostream>
#include <math.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
__global__
void vecadd(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}
 
int main(void)
{
  int N = 1<<20;
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  int Ndef;
  scanf("%d",&Ndef);
  for (int i = 0; i < Ndef; i++) 
  {
    scanf("%f%f",&x[i],&y[i]);
  }
 
  vecadd<<<1, 256>>>(Ndef, x, y);
  hipDeviceSynchronize();
   for (int i = 0; i < Ndef; i++) 
  {
    printf("%f ",y[i]);
  }
  hipFree(x);
  hipFree(y);
  
  return 0;
}
