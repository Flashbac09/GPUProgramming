#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
//vector add, given dimension n.
//1 block,n threads.
#define n 2
double a[n],b[n],c[n];
//KEY:address for memcpy,set pointers.//
double *dev_a,*dev_b,*dev_c;
__host__ int main()
{
    //step 0 data
    for(int i=0;i<n;i++)
    scanf("%lf%lf",&a[i],&b[i]);
    //step 1 malloc for device
    hipMalloc((void**)&dev_a,n*sizeof(double));
    hipMalloc((void**)&dev_b,n*sizeof(double));
    hipMalloc((void**)&dev_c,n*sizeof(double));
    //step 2 data that device needs(H2D)
    hipMemcpy(dev_a,a,N*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,N*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dev_c,c,N*sizeof(double),hipMemcpyHostToDevice);
    //step 3 kernel function
    vecadd<<<1,n>>>(dev_a,dev_b,dev_c);
    //step 4 send result back to host(D2H)
    hipMemcpy(c,dev_c,N*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<n;i++)
    {
        printf("%lf + %lf = %lf\n",a[i],b[i],c[i]);
    }
    //step end:CUDAFREE
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}

__global__ void vecadd(double *a,double *b,double *c)
{
    if(threadIdx.x<n)
    c[threadIdx.x]=a[threadIdx.x]+b[threadIdx.x];
}