#include<iostream>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<string.h>
//memcpy example
__global__ void simpleadd(int a, int b, int *c)
{
    *c=a+b;
    printf("dev:%d\n",*c);
}

int main()
{
    //step 0
    int c;
    int *c4dev;
    //step 1 malloc
    hipMalloc((void**)&c4dev,sizeof(int));
    //step 2 kernel function
    simpleadd<<<2,19>>>(8,6,c4dev);
    //step 3 memcpy(d2h)
    hipMemcpy(&c,c4dev,sizeof(int),hipMemcpyDeviceToHost);
    printf("Host:%d\n",c);
    hipFree(c4dev);
    return 0;
}